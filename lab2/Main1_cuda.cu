#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Main1_cuda.cuh"
#include "cuPrintf.cu"

//since we can't really dynamically size this array,
//let's leave its size at the default polynomial order
__constant__ float constant_c[10];


__global__
void
cudaSum_atomic_kernel(const float* const inputs,
                                     unsigned int numberOfInputs,
                                     const float* const c,
                                     unsigned int polynomialOrder,
                                     float* output) {
    
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    float partial_sum = 0.0;
    
    while (index < numberOfInputs) {
        // calculate polynomial value at inputs[index]
        float r = inputs[index];
        float result = 0.0;
        float power = 1.0;

        for (unsigned int i = 0; i < polynomialOrder; i++) {
            result += (c[i] * power);
            power *= r;
        }

        partial_sum += result; // add result of P(r) to partial_sum

        // increment index to next value, if this thread has to handle
        // multiple elements
        index += blockDim.x * gridDim.x;
    }
    atomicAdd(output, partial_sum);
    

}

__global__
void
cudaSum_linear_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs, 
                                  const float* const c,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    
    
    extern __shared__ float partial_outputs[];

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    float partial_sum = 0.0;
    
    while (index < numberOfInputs) {

        // calculate polynomial value at inputs[index]
        float r = inputs[index];
        float result = 0.0;
        float power = 1.0;

        for (unsigned int i = 0; i < polynomialOrder; i++) {
            result += (c[i] * power);
            power *= r;
        }

        partial_sum += result; // add result of P(r) to partial_sum

        index += blockDim.x * gridDim.x; // increment index to next value
    }
    partial_outputs[threadIdx.x] = partial_sum;

    // Make all threads in the block finish before computing
    syncthreads(); 

    // Here, start with the first thread's partial sum.  Add the rest of the threads'
    // partial sums, so that partial_sum contains the sum from all of threads
    // of the block. 
    if (threadIdx.x == 0) {
        for (unsigned int threadIndex = 1; threadIndex < blockDim.x; threadIndex++) {
            partial_sum += partial_outputs[threadIndex];
        }

        // Now, finally accumulate (we've already added together all the partial sums)
        // Note that atomicAdd is called is once per block now, instead of per thread
        atomicAdd(output, partial_sum);
    }    
}
 

/* Used in Assignment 2. Coming soon! */
__global__
void
cudaSum_divtree_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs, 
                                  const float* const c,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    
    extern __shared__ float partial_outputs[];

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    float partial_sum = 0.0;
    
    while (index < numberOfInputs) {

        // calculate polynomial value at inputs[index]
        float r = inputs[index];
        float result = 0.0;
        float power = 1.0;

        for (unsigned int i = 0; i < polynomialOrder; i++) {
            result += (c[i] * power);
            power *= r;
        }

        partial_sum += result; // add result of P(r) to partial_sum

        index += blockDim.x * gridDim.x; // increment index to next value
    }
    partial_outputs[threadIdx.x] = partial_sum;


    // Make all threads in the block finish before computing
    syncthreads(); 

    

    cuPrintf("p[%d] = %f \n", threadIdx.x, partial_outputs[threadIdx.x]);

    
    int offset = 1;
    cuPrintf("%d \n", offset < blockDim.x);
    
    while (offset < blockDim.x) {

        if (threadIdx.x == 0) {
            cuPrintf("OFFSET: %d \n", offset);
            cuPrintf("MOD: %d \n", threadIdx.x % offset == 0);
        }
        
        if (threadIdx.x % offset == 0) {
            float add = partial_outputs[(int)threadIdx.x + offset];
            float current = partial_outputs[(int)threadIdx.x];
            //cuPrintf("add: %f   current: %f \n", add, current);
            cuPrintf("ADD: %d \n", (int)threadIdx.x);
            //partial_outputs[(int)threadIdx.x] = 1.0;//add + current;//add + current;
            //partial_outputs[threadIdx.x] = add;
            partial_outputs[threadIdx.x] = partial_outputs[threadIdx.x] + partial_outputs[threadIdx.x + offset];
        }
        
        offset = offset * 2;
        syncthreads();
    }
    

    //cuPrintf("PP[%d] = %f \n", threadIdx.x, partial_outputs[threadIdx.x]);

    /*

    cuPrintf("p[0] = %d \n", partial_outputs[0]);

    if (threadIdx.x == 0) {
        atomicAdd(output, partial_outputs[0]);
    }
    */
    /*
    // Here, start with the first thread's partial sum.  Add the rest of the threads'
    // partial sums, so that partial_sum contains the sum from all of threads
    // of the block. 
    if (threadIdx.x == 0) {
        for (unsigned int threadIndex = 1; threadIndex < blockDim.x; threadIndex++) {
            partial_sum += partial_outputs[threadIndex];
        }

        // Now, finally accumulate (we've already added together all the partial sums)
        // Note that atomicAdd is called is once per block now, instead of per thread
        atomicAdd(output, partial_sum);
    }   
    */ 
}

/* Used in Assignment 2. Coming soon! */
__global__
void
cudaSum_nondivtree_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs, 
                                  const float* const c,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    

}

/* Used in Assignment 2. Coming soon! */
__global__
void
cudaSum_constmem_kernel(const float* const inputs, 
                                  unsigned int numberOfInputs,
                                  unsigned int polynomialOrder, 
                                  float * output) {
    

}




void
cudaSumPolynomials(const float* const input,
                            const size_t numberOfInputs,
                            const float* const c,
                            const size_t polynomialOrder,
                            const Style style,
                            const unsigned int maxBlocks,
                            float * const output) {


    //Input values (your "r" values) go here on the GPU
    float *dev_input;
    
    //Your polynomial coefficients go here (GPU)
    float *dev_c;
    
    //Your output will go here (GPU)
    float *dev_output;
    
    // Allocate memory on the GPU for our inputs
    hipMalloc((void **) &dev_input, numberOfInputs*sizeof(float));
    hipMemcpy(dev_input, input, numberOfInputs*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **) &dev_c, polynomialOrder*sizeof(float));
    hipMemcpy(dev_c, c, polynomialOrder*sizeof(float), hipMemcpyHostToDevice);

    // Allocate memory on the GPU for outputs
    hipMalloc((void **) &dev_output, sizeof(float));
    hipMemset(dev_output, 0, sizeof(float)); // make sure to initialize output to 0!

    cudaPrintfInit();    
    
    const unsigned int threadsPerBlock = 512;
    const unsigned int blocks 
                = min((float)maxBlocks, 
                        ceil(numberOfInputs/(float)threadsPerBlock));

    if (style == mutex) {
        cudaSum_atomic_kernel<<<blocks, threadsPerBlock>>>(
                dev_input, numberOfInputs, dev_c, polynomialOrder, dev_output);
    } else if (style == linear) {
        cudaSum_linear_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                dev_c, polynomialOrder, dev_output);
    } else if (style == divtree) {
        cudaSum_divtree_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                dev_c, polynomialOrder, dev_output);
    } else if (style == nondivtree) {
        cudaSum_nondivtree_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                dev_c, polynomialOrder, dev_output);
    } else if (style == constmem) {
        
        //initialize the constant memory
        hipMemcpyToSymbol(HIP_SYMBOL("constant_c"), c, polynomialOrder * sizeof(float),
                0, hipMemcpyHostToDevice);
        
        cudaSum_constmem_kernel<<<blocks, threadsPerBlock, 
                threadsPerBlock*sizeof(float)>>>(dev_input, numberOfInputs, 
                polynomialOrder, dev_output);
    } else {
        printf("Unknown style\n");
    }

    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    // Copy output from device to host
    hipMemcpy(output, dev_output, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(dev_input);
    hipFree(dev_c);
    hipFree(dev_output);
}
